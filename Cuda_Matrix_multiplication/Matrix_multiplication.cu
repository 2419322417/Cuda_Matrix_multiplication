#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <math.h>

#define TILE_WIDTH 16
#define STREAMS 4      

//Kernel������ִ�о���˷�
__global__ void Matrix_MulKernel(int m, int n, int k, float* A, float* B, float* C) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < m && Col < k) {
        float cvalue = 0.0f;
        for (int i = 0; i < n; i++) {
            cvalue += A[Row * n + i] * B[i * k + Col];
        }
        C[Row * k + Col] = cvalue;
    }
}

//��һ���Ż���ʹ�ù����ڴ��ƽ���㷨�Ż��ľ���˷�Kernel
__global__ void Matrix_MulKernel_Tiled(int m, int n, int k, float* A, float* B, float* C) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int num_tiles = (n + TILE_WIDTH - 1) / TILE_WIDTH;
    float Cvalue = 0.0f;

    for (int t = 0; t < num_tiles; ++t) {
        // ����A��tile
        int a_row = Row;
        int a_col = t * TILE_WIDTH + threadIdx.x;
        int b_row = t * TILE_WIDTH + threadIdx.y;
        int b_col = Col;
        if (a_row < m && a_col < n) {
            ds_A[threadIdx.y][threadIdx.x] = A[a_row * n + a_col];
        }
        else {
            ds_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // ����B��tile (ת�ü�����ʵ�ֺϲ�����)
        if (b_row < n && b_col < k) {
            ds_B[threadIdx.y][threadIdx.x] = B[b_row * k + b_col];
        }
        else {
            ds_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // �ۻ��ڻ�
        for (int i = 0; i < TILE_WIDTH; ++i) {
            Cvalue += ds_A[threadIdx.y][i] * ds_B[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (Row < m && Col < k) {
        C[Row * k + Col] = Cvalue;
    }
}

//�ڶ����Ż����ڹ����ڴ������padding������Bank Conflict��Kernel
__global__ void Matrix_MulKernel_Tiled_Padding(int m, int n, int k, float* A, float* B, float* C) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH + 1];  // +1����Bank Conflict
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH + 1];

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    float Cvalue = 0.0f;

    for (int t = 0; t < (n - 1) / TILE_WIDTH + 1; ++t) {
        if (Row < m && t * TILE_WIDTH + threadIdx.x < n) {
            ds_A[threadIdx.y][threadIdx.x] = A[Row * n + t * TILE_WIDTH + threadIdx.x];
        }
        else {
            ds_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (t * TILE_WIDTH + threadIdx.y < n && Col < k) {
            ds_B[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * k + Col];
        }
        else {
            ds_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            Cvalue += ds_A[threadIdx.y][i] * ds_B[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (Row < m && Col < k) {
        C[Row * k + Col] = Cvalue;
    }
}

//�������Ż���ֱ�ӵ���block��С��

//�������Ż����Ĵ����Ż�
__global__ void Matrix_MulKernel_RegTiling(int m, int n, int k, float* A, float* B, float* C) {
    __shared__ float d_B[TILE_WIDTH][TILE_WIDTH];

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    float Cvalue = 0.0f;

    for (int t = 0; t < (n - 1) / TILE_WIDTH + 1; ++t) {
        // ���� B �� tile �������ڴ�
        if (t * TILE_WIDTH + threadIdx.y < n && Col < k) {
            d_B[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * k + Col];
        }
        else {
            d_B[threadIdx.y][threadIdx.x] = 0.0f;
        }
        __syncthreads();

        #pragma unroll//չ��ѭ���Լ��ٿ��ƿ���
        for (int i = 0; i < TILE_WIDTH; ++i) {
            int a_col = t * TILE_WIDTH + i;
            float a = 0.0f;
            if (Row < m && a_col < n) {
                a = A[Row * n + a_col];
            }
            Cvalue += a * d_B[i][threadIdx.x];
        }
        __syncthreads();
    }

    if (Row < m && Col < k) {
        C[Row * k + Col] = Cvalue;
    }
}

//�������Ż���ʹ������Streams��ʵ���ص���������ݴ��䡣
void Matrix_Mul_Overlapping(int m, int n, int k, float* h_A, float* h_B, float* h_C) {
    int rowsPerStream = m / STREAMS;
    int remainder = m % STREAMS;

    float* d_A[STREAMS];
    float* d_C[STREAMS];
    hipStream_t streams[STREAMS];

    // ֻ����һ�� d_B
    float* d_B;
    size_t size_B = (size_t)n * k * sizeof(float);
    hipMalloc(&d_B, size_B);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    for (int i = 0; i < STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
        int thisRows = rowsPerStream + (i < remainder ? 1 : 0);
        if (thisRows == 0) continue;

        size_t size_A = (size_t)thisRows * n * sizeof(float);
        size_t size_C = (size_t)thisRows * k * sizeof(float);
        hipMalloc(&d_A[i], size_A);
        hipMalloc(&d_C[i], size_C);
    }

    dim3 threads(TILE_WIDTH, TILE_WIDTH);

    // ��һ��ѭ�������������ݴ�������HtoD��������Ե�����
    int rowOffset = 0;
    for (int i = 0; i < STREAMS; ++i) {
        int thisRows = rowsPerStream + (i < remainder ? 1 : 0);
        if (thisRows == 0) continue;
        size_t size_A = (size_t)thisRows * n * sizeof(float);
        hipMemcpyAsync(d_A[i], h_A + (size_t)rowOffset * n, size_A, hipMemcpyHostToDevice, streams[i]);
        rowOffset += thisRows;
    }

    // �ڶ���ѭ���������м������������Ե�����
    rowOffset = 0;
    for (int i = 0; i < STREAMS; ++i) {
        int thisRows = rowsPerStream + (i < remainder ? 1 : 0);
        if (thisRows == 0) continue;
        dim3 blocks((k + TILE_WIDTH - 1) / TILE_WIDTH, (thisRows + TILE_WIDTH - 1) / TILE_WIDTH);
        Matrix_MulKernel_Tiled << < blocks, threads, 0, streams[i] >> > (thisRows, n, k, d_A[i], d_B, d_C[i]);
        rowOffset += thisRows;
    }

    // ������ѭ���������н����������DtoH��������Ե�����
    rowOffset = 0;
    for (int i = 0; i < STREAMS; ++i) {
        int thisRows = rowsPerStream + (i < remainder ? 1 : 0);
        if (thisRows == 0) continue;
        size_t size_C = (size_t)thisRows * k * sizeof(float);
        hipMemcpyAsync(h_C + (size_t)rowOffset * k, d_C[i], size_C, hipMemcpyDeviceToHost, streams[i]);
        rowOffset += thisRows;
    }


    // �ȴ����� stream ���
    hipDeviceSynchronize();

    // ����
    for (int i = 0; i < STREAMS; ++i) {
        // ���ָ���Ƿ��ѷ��䣬������ thisRows == 0 ������³���
        if (d_A[i]) hipFree(d_A[i]);
        if (d_C[i]) hipFree(d_C[i]);
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_B);
}

//CPU�汾����˷���������֤���
void Matrix_MulCPU(int m, int n, int k, float* A, float* B, float* C) {
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < k; col++) {
            float value = 0.0f;
            for (int i = 0; i < n; i++) {
                value += A[row * n + i] * B[i * k + col];
            }
            C[row * k + col] = value;
        }
    }
}

int Compare_Results(int m, int k, float* C_cpu, float* C_gpu) {
    float eps = 1e-5f;
    for (int i = 0; i < m * k; i++) {
        if (fabsf(C_cpu[i] - C_gpu[i]) > eps) {
            return 0; 
        }
    }
    return 1; 
}

int main() {
    int m = 512; 
    int n = 512; 
    int k = 512; 

	//���������ڴ�
    size_t size_A = m * n * sizeof(float);
    size_t size_B = n * k * sizeof(float);
    size_t size_C = m * k * sizeof(float);

    float* h_A = nullptr;
    float* h_B = nullptr;
    float* h_C = nullptr;
    float* h_C_cpu = (float*)malloc(size_C);
    
     // ��ͨ malloc������ǰ�����Ż�������
     h_A = (float*)malloc(size_A);
     h_B = (float*)malloc(size_B);
     h_C = (float*)malloc(size_C);

     // pinned memory
     /*hipHostAlloc((void**)&h_A, size_A, hipHostMallocDefault);
     hipHostAlloc((void**)&h_B, size_B, hipHostMallocDefault);
     hipHostAlloc((void**)&h_C, size_C, hipHostMallocDefault);*/

	//��ʼ������A��B,ͬʱ��֤���ÿ��Ԫ�ض�Ϊ2.0f*n
    for (int i = 0; i < m * n; i++) h_A[i] = 1.0f; 
    for (int i = 0; i < n * k; i++) h_B[i] = 2.0f;
    for (int i = 0; i < m * k; i++) h_C[i] = 0.0f;

	//�����豸�ڴ�
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    /*dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (m + threadsPerBlock.y - 1) / threadsPerBlock.y);
    Matrix_MulKernel <<< numBlocks, threadsPerBlock >>> (m, n, k, d_A, d_B, d_C);*/

    dim3 threadsPerBlock_tiled(TILE_WIDTH, TILE_WIDTH);
    dim3 numBlocks_tiled((k + TILE_WIDTH - 1) / TILE_WIDTH,
        (m + TILE_WIDTH - 1) / TILE_WIDTH);
    Matrix_MulKernel_Tiled <<< numBlocks_tiled, threadsPerBlock_tiled >>> (m, n, k, d_A, d_B, d_C);
    /*Matrix_MulKernel_Tiled_Padding<<<numBlocks_tiled, threadsPerBlock_tiled >>>(m, n, k, d_A, d_B, d_C);
    Matrix_MulKernel_RegTiling<<<numBlocks_tiled, threadsPerBlock_tiled>>>(m, n, k, d_A, d_B, d_C);
    Matrix_Mul_Overlapping(m, n, k, h_A, h_B, h_C);*/

	//ͬ���ȴ�GPU���
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    //printf(" C (m=%d, k=%d):\n", m, k);
    //for (int i = 0; i < m; i++) {
    //    for (int j = 0; j < k; j++) {
    //        printf("%5.1f ", h_C[i * k + j]);
    //    }
    //    printf("\n");
    //}

    //CPU����
    Matrix_MulCPU(m, n, k, h_A, h_B, h_C_cpu);

    //��֤���
    if (Compare_Results(m, k, h_C_cpu, h_C)) {
        printf(" \nCPU and GPU results match.\n");
    }
    else {
        printf(" \nCPU and GPU results differ.\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_C_cpu);
    //ǰ�����Ż�����
    free(h_A);
    free(h_B);
    free(h_C);

    //��Ӧpinned�ͷ�

    /*hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);*/

    return 0;
}
