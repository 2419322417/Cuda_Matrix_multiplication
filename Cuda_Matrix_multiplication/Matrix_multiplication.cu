#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>

//Kernel������ִ�о���˷�
__global__ void MatrixMulKernel(int m, int n, int k, float* A, float* B, float* C) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < m && Col < k) {
        float Cvalue = 0.0f;
        for (int i = 0; i < n; i++) {
            Cvalue += A[Row * n + i] * B[i * k + Col];
        }
        C[Row * k + Col] = Cvalue;
    }
}

int main() {
    int m = 4; 
    int n = 3; 
    int k = 5; 

	//���������ڴ�
    size_t size_A = m * n * sizeof(float);
    size_t size_B = n * k * sizeof(float);
    size_t size_C = m * k * sizeof(float);

    float* h_A = (float*)malloc(size_A);
    float* h_B = (float*)malloc(size_B);
    float* h_C = (float*)malloc(size_C);

	//��ʼ������A��B,ͬʱ��֤���ÿ��Ԫ�ض�Ϊ2.0f*n=6.0f
    for (int i = 0; i < m * n; i++) h_A[i] = 1.0f; 
    for (int i = 0; i < n * k; i++) h_B[i] = 2.0f;

	//�����豸�ڴ�
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    MatrixMulKernel <<<numBlocks,threadsPerBlock>>> (m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    printf("Result matrix C (m=%d, k=%d):\n", m, k);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            printf("%5.1f ", h_C[i * k + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
